#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

int main() {
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
